#include <iostream>


#include "raytracingCommon.h"

#ifdef __HIPCC__
#define CUDA_CALLABLE __host__ __device__
#else
#define CUDA_CALLABLE
#endif



int main() {
    std::cout << "Hello, World!" << std::endl;

    // World
    hittableList world;
    world.add(make_shared<sphere>(point3(0.f,0.f,-1.f),0.5));
    world.add(make_shared<sphere>(point3(0.f,-100.5f,-1.f), 100.f));

    // Camera
    camera cam;
    // cam.aspectRatio = 16.0/9.0;
    cam.aspectRatio = 2.0/1.0;
    cam.imageWidth = IMAGEWIDTH;
    cam.samplePerPixel = MAXSPP;
    cameraInfo camCuda = cam.toCameraInfo();
    printf("Camera pss:%f\n",camCuda.pixelSampleScale);
    cam.render("image.ppm",world);
    printf("------------------------");
    render(camCuda,"imageCuda.ppm",world.toSphserList());
    return 0;
}
